#include "hip/hip_runtime.h"
__global__
void bitonic_sort_step(int length, long * arr, int j, int k, bool dir)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i< length; i+=stride)
  {
    int ixj = i^j;
    if((ixj)>i) {
      if(dir == true)
      {      
        if((i&k)==0)
        {
          if(arr[i]>arr[ixj] && ixj<length)
          {
            long tmp = arr[i];
            arr[i] = arr[ixj];
            arr[ixj] = tmp;

          }
        }
        if((i&k)!=0)
        {
          if(arr[i]<=arr[ixj] && ixj<length)
          {
            long tmp = arr[i];
            arr[i] = arr[ixj];
            arr[ixj] = tmp;
          }        
        }
      }
      else
      {      
        if((i&k)!=0)
        {
          if(arr[i]>arr[ixj] && ixj<length)
          {
            long tmp = arr[i];
            arr[i] = arr[ixj];
            arr[ixj] = tmp;
            tmp = arr[length + i];
            arr[ROW1*length + i] = arr[ROW1*length + ixj];
            arr[ROW1*length + ixj] = tmp; 
          }
          else if(arr[i]==arr[ixj]){
            if(arr[ROW1*length + i]>arr[ROW1*length + ixj]){
              long tmp = arr[i];
              arr[i] = arr[ixj];
              arr[ixj] = tmp;
              tmp = arr[ROW1*length + i];
              arr[ROW1*length + i] = arr[ROW1*length + ixj];
              arr[ROW1*length + ixj] = tmp;  
            }
          }
        }
        if((i&k)==0)
        {
          if(arr[i]<=arr[ixj] && ixj<length)
          {
            long tmp = arr[i];
            arr[i] = arr[ixj];
            arr[ixj] = tmp;
            tmp = arr[ROW1*length + i];
            arr[ROW1*length + i] = arr[ROW1*length + ixj];
            arr[ROW1*length + ixj] = tmp;
          }
          else if(arr[i]==arr[ixj]){
            if(arr[ROW1*length + i]<arr[ROW1*length + ixj]){
              long tmp = arr[i];
              arr[i] = arr[ixj];
              arr[ixj] = tmp;
              tmp = arr[ROW1*length + i];
              arr[ROW1*length + i] = arr[ROW1*length + ixj];
              arr[ROW1*length + ixj] = tmp;      
              
            }
          }        
        }
      }
    }
  }
}

long * bitonic_sort(int length, int numBlock, long * arr)
{
  int nextP2 = length == 1 ? 1 : 1 << (32 - __lzcnt(length-1));
  long* cudaArr;
  hipMallocManaged(&cudaArr, length*ROW2*sizeof(long));
  hipMemcpy(cudaArr, arr, length*ROW2*sizeof(long), hipMemcpyHostToDevice);
  for(int i=2; i<=nextP2; i=i<<1) {
    for(int j=i>>1; j>0; j=j>>1){
      int tmp = length - 1;
      int tmpxj = (tmp^j);
      bool accending = true;
      if(tmpxj>tmp){
        int dir = (i&tmp);
        if(dir != 0) accending = false;
      }
      bitonic_sort_step<<<numBlock, BLOCKSIZE>>>(length, cudaArr, j, i, accending);
      hipDeviceSynchronize();
    }
  }
  return cudaArr;
}
